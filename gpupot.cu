#include "hip/hip_runtime.h"
// Programer: Pablo Bena Llambay / Mark Gieles
// Date: 25/06/2012
// A short example: calculation of potential of many particles.
// Compilation: write nvcc potential.cu

#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "mameclot.h"

#define BLOCKSIZE 256
	
//Computing the Potential on the Device
__global__ void compute_potential_gpu(float *m, float *x, float *y, float *z, float *pot,int N) {
  int i,j; 
  float rij, rijx, rijy, rijz;
  float xi, yi, zi;
  float potential; // Be careful with float or singles variables!
  i = threadIdx.x + blockIdx.x*blockDim.x;
  if (i<N) {
    xi = x[i];
    yi = y[i];
    zi = z[i];
    potential = 0.0f;
    for (j=0; j<N; j++) {
      rijx = xi - x[j];
      rijy = yi - y[j];
      rijz = zi - z[j];
      if (j!=i){
      	 rij = -m[j]/sqrt(rijx*rijx + rijy*rijy + rijz*rijz);
	 potential += rij;
	 }	 	   
    }
    pot[i] = potential;
  }
}


extern "C" void calculate_potential(float *m, float *x, float *y, float *z, float *pot, int *np) 
{
  float *m_d,*x_d,*y_d,*z_d,*pot_d; // Device variables!

  int N=*np;

  //Allocating memory on the Device
  hipMalloc(&m_d  , sizeof(float)*N); 

  hipMalloc(&x_d  , sizeof(double)*N); 
  hipMalloc(&y_d  , sizeof(float)*N);
  hipMalloc(&z_d  , sizeof(float)*N); 
  hipMalloc(&pot_d, sizeof(float)*N);

  hipMemcpy(m_d,m    , sizeof(float)*N, hipMemcpyHostToDevice); // Host -> Device
  hipMemcpy(x_d,x    , sizeof(float)*N, hipMemcpyHostToDevice); // Host -> Device
  hipMemcpy(y_d,y    , sizeof(float)*N, hipMemcpyHostToDevice); // Host -> Device
  hipMemcpy(z_d,z    , sizeof(float)*N, hipMemcpyHostToDevice); // Host -> Device
  hipMemcpy(pot_d,pot, sizeof(float)*N, hipMemcpyHostToDevice); // Host -> Device

  
  compute_potential_gpu <<<((N+BLOCKSIZE-1))/BLOCKSIZE,BLOCKSIZE >>>(m_d,x_d, y_d, z_d, pot_d,N);
  hipMemcpy(pot,pot_d, sizeof(float)*N, hipMemcpyDeviceToHost); // Host -> Device
    
  //Freeing memory
  hipFree(m_d);
  hipFree(x_d);
  hipFree(y_d);
  hipFree(z_d);
  hipFree(pot_d);
  
  return;
}
